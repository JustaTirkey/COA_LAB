// this program computed matrix multipication on the gpu using cuda

#include <hip/hip_runtime.h>
#include <cstdlib>
#include <cassert>
#include <bits/stdc++.h>
using namespace std;

__global__ void matMult(int *a, int *b, int *c,int N){
    // calculate the global row and column for each thread
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    // boundary check for our matrix
    if(row<N && col<N){
        int tmp =0;
        for(int i=0;i<N;i++){
            tmp +=a[row*N+i]* b[i*N+col];
        }
        // write back the result
        c[row*N+ col]= tmp;
        // cout<<tmp<<" ";
    }
}


// verify the result in cpu
void verify(int *a, int *b, int *c,int N){
    for(int i=0;i<N;i++){
        for(int j=0;j<N;j++){
            int sum=0;
            for(int k=0;k<N;k++){
                sum+=a[i*N+k]*b[k*N+j];
            }
            // verify the tmp is correct
            // assert(sum == c[i*N + j]);
            // assert(sum == c[i * N + j]);
            // cout<<sum<<" ";
        }
    }
}

int main()
{
    //set our square matrix dimension (2^10 * 2^10 default)
    int N = 1<<10;
    size_t bytes = N*N * sizeof(int);

    // allocate memory for the matrices
    int * a, *b, *c;
    hipHostMalloc(&a, bytes, hipHostMallocDefault);
    hipHostMalloc(&b, bytes, hipHostMallocDefault);
    hipHostMalloc(&c, bytes, hipHostMallocDefault);

    // input the matrices for random number
    for(int i=0;i<N*N;i++){
        a[i]= rand() % 10;
        cout<<a[i]<<" ";
    }
    for(int i=0;i<N*N;i++){
        b[i]= rand() % 10;
    }

    // set our block and grid dimensions
    int threads = 16; // 16 threads per thread block
    int blocks = (N * threads - 1) / threads;

    // setup our kernel launch parameters
    // dim3 is just a structure designed for storing block and grid dimensions.
    dim3 THREADS(threads , threads);
    dim3 BLOCKS(blocks, blocks);

    // launch our kernel
    matMult<<<BLOCKS, THREADS>>>(a,b,c,N);
    hipDeviceSynchronize();
    // cudaThreadSynchronize();
    
    // verify the result
    verify(a,b,c,N);
    cout<<"Program completed successfully\n";
    return 0;
}